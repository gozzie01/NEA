#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <fstream>
#include <nlohmann/json.hpp>
#include <cstring>
#include <direct.h>

// parents evening timetabler
void shuffleSame(int *array1, int *array2, size_t n)
{
	if (n > 1)
	{
		// set the seed
		srand(time(NULL));
		size_t i;
		for (i = 0; i < n - 1; i++)
		{
			size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
			int t1 = array1[j];
			int t2 = array2[j];
			array1[j] = array1[i];
			array2[j] = array2[i];
			array1[i] = t1;
			array2[i] = t2;
		}
	}
}

__global__ void initRand(hiprandState *state, unsigned long seed)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed + id, id, 0, &state[id]);
}

__global__ void bundledFunction(hiprandState *state, int *appTimes, int *teachers, int *teacherMin, int *teacherMax, int numTeachers, int *parents, int *parentMin, int *parentMax, int numParents, int numSlots, int numThreads, int numAppointments, int *teacherTimes, int *parentTimes, int maxTeacher, int maxParent, int *fitnesses, int startIndex)
{
	const unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= numThreads)
	{
		return; // if the thread is out of bounds return
	}
	__shared__ int fitness[4096];
	if (fitness[id] != 0)
	{
		fitness[id] = 0;
	}
	// print numAppointments
	// print the parents array
	for (int currentMax = startIndex; currentMax < numAppointments; currentMax++)
	{
		// set all times to -1
		for (int i = 0; i < numSlots; i++)
		{
			teacherTimes[id * numSlots + i] = -1;
			parentTimes[id * numSlots + i] = -1;
		}
		for (int i = 0; i < currentMax; i++)
		{
			if ((teachers[i] == teachers[currentMax]) && (appTimes[id * numAppointments + i] != -2))
			{
				int j = -1;
				do
				{
					j++;
				} while (teacherTimes[id * numSlots + j] != -1);
				teacherTimes[id * numSlots + j] = appTimes[id * numAppointments + i];
			}

			if ((parents[i] == parents[currentMax]) && (appTimes[id * numAppointments + i] != -2))
			{
				int j = -1;
				do
				{
					j++;
				} while ((parentTimes[id * numSlots + j] != -1));
				parentTimes[id * numSlots + j] = appTimes[id * numAppointments + i];
			}
		}

		bool valid = true;
		int gMin = 0;
		int gMax = numSlots;

		// range between the highest min and lowest max

		if (teacherMin[teachers[currentMax]] > parentMin[parents[currentMax]])
		{
			gMin = teacherMin[teachers[currentMax]];
		}
		else
		{
			gMin = parentMin[parents[currentMax]];
		}
		if (teacherMax[teachers[currentMax]] < parentMax[parents[currentMax]])
		{
			gMax = teacherMax[teachers[currentMax]];
		}
		else
		{
			gMax = parentMax[parents[currentMax]];
		}
		int range = gMax - gMin;
		//if id is 0 print the range
		for (int i = 0; i < range; i++)
		{
			valid = true; // Reset valid to true at the start of each iteration
			// check that the teacher is free
			bool teacherFree = true;
			for (int j = 0; j < numSlots; j++)
			{
				if (teacherTimes[id * numSlots + j] == i + gMin)
				{
					teacherFree = false;
					break; // No need to check further if teacher is not free
				}
			}

			// check that the parent is free
			bool parentFree = true;
			for (int j = 0; j < numSlots; j++)
			{
				if (parentTimes[id * numSlots + j] == i + gMin)
				{
					parentFree = false;
					break; // No need to check further if parent is not free
				}
			}

			if (!(teacherFree && parentFree))
			{
				valid = false;
			}

			if (valid) // If valid is true, no need to check further time slots
			{
				break;
			}
		}
		//  if the appointment is valid loop until a valid time is found
		if (valid)
		{
			while (true)
			{

				// generate a random time between the highest min and lowest max
				int time = hiprand(&state[id]) % range + gMin;
				// check that the teacher is free
				bool teacherFree = true;
				for (int j = 0; j < numSlots; j++)
				{
					if (teacherTimes[id * numSlots + j] == time)
					{
						teacherFree = false;
					}
				}
				// check that the parent is free
				bool parentFree = true;
				for (int j = 0; j < numSlots; j++)
				{
					if (parentTimes[id * numSlots + j] == time)
					{
						parentFree = false;
					}
				}
				if (teacherFree && parentFree)
				{
					appTimes[id * numAppointments + currentMax] = time;
					break;
				}
			}
		}
		else
		{
			// if the appointment is not valid set the time to -2
			appTimes[id * numAppointments + currentMax] = -2;
		}
	}

	for (int i = 0; i < numAppointments; i++)
	{
		// if the appointment is not valid
		if (appTimes[id * numAppointments + i] == -2)
		{
			fitness[id] -= 10;
			continue;
		}

		for (int j = i + 1; j < numAppointments; j++)
		{
			// if appointment is invalid
			if (appTimes[id * numAppointments + j] == -2)
			{
				continue;
			}
			if ((parents[i] != 0) && (parents[i] == parents[j]))
			{
				// if appointments are on the same slot and with the same parent
				if (appTimes[id * numAppointments + i] == appTimes[id * numAppointments + j])
				{
					fitness[id] -= 6;
				}
				// if appointments have 1 slot in between
				if (abs(appTimes[id * numAppointments + i] - appTimes[id * numAppointments + j]) == 2)
				{
					fitness[id] += 3;
				}
				// if appointments have 2 slots in between
				if (abs(appTimes[id * numAppointments + i] - appTimes[id * numAppointments + j]) == 3)
				{
					fitness[id] += 1;
				}
				// if they are consecutive slots
				if (abs(appTimes[id * numAppointments + i] == appTimes[id * numAppointments + j]) == 1)
				{
					fitness[id] -= 1;
				}
			}
		}
	}
	fitnesses[id] = fitness[id];
}

int main()
{
	char cwd[FILENAME_MAX];
	if (_getcwd(cwd, sizeof(cwd)) != NULL) {
		std::cout << "Current working directory: " << cwd << std::endl;
	}
	else {
		std::cerr << "getcwd() error" << std::endl;
		return 1;
	}
	const int numThreads = 4096;
	const int threadsPerBlock = 256;
	const int iterations = 4096;
	nlohmann::json js;
	//print current directory
	

	// Variables related to file input
	try {
		std::ifstream file("input.json");
		file >> js;
	}
	catch (std::exception &e) {
		std::cerr << "Error: " << e.what() << std::endl;
		return 1;
	}

	// Variables related to appointments
	const int numParents = js["parents"].size() + 1;
	const int numSlots = js["duration"];
	const int numTeachers = js["teachers"].size() + 1;
	const int numAppointments = js["appointments"].size();
	const int numBlocked = js["blockedTeacher"].size() + js["blockedParent"].size();
	const int wantedAppointments = js["wantedAppointments"].size();
	printf("Num Wantged: %d\n", wantedAppointments);

	// Variables related to CUDA
	const int blocksPerGrid = (numThreads + threadsPerBlock - 1) / threadsPerBlock;

	// arrays
	int *teachers = (int *)malloc((numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2);
	int *parents = (int *)malloc((numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2);
	int *teacherMin = (int *)malloc((numTeachers + 1) * sizeof(int) * 2);
	int *teacherMax = (int *)malloc((numTeachers + 1) * sizeof(int) * 2);
	int *parentMin = (int *)malloc((numParents + 1) * sizeof(int) * 2);
	int *parentMax = (int *)malloc((numParents + 1) * sizeof(int) * 2);
	int *teacherMap = (int *)malloc((numTeachers + 1) * sizeof(int) * 2);
	int *parentMap = (int *)malloc((numParents + 1) * sizeof(int) * 2);
	int *singleTimes = (int *)malloc((numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2);
	int *appTimes = (int *)malloc((numAppointments + numBlocked + wantedAppointments) * numThreads * sizeof(int) * 2);
	int *fitness = (int *)malloc(numThreads * sizeof(int) * 2);
	int *bestTimes = (int *)malloc(iterations * (numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2 * 2);
	int *bestFitnesses = (int *)malloc(iterations * 2 * sizeof(int) * 2);


	// variables to store the order of appointments for previous iterations
	int *teachersStore = (int*)malloc((numAppointments + numBlocked + wantedAppointments) * iterations * sizeof(int) * 2);
	int *parentsStore = (int*)malloc((numAppointments + numBlocked + wantedAppointments) * iterations * sizeof(int) * 2);

	// Initialize maps
	parentMap[0] = 0;
	teacherMap[0] = 0;

	for (int i = 0; i < numTeachers; i++)
	{
		teacherMin[i] = 0;
		teacherMax[i] = numSlots - 1;
	}
	for (int i = 0; i < numParents; i++)
	{
		parentMin[i] = 0;
		parentMax[i] = numSlots - 1;
	}
	// fill teacher and parent maps
	for (int i = 0; i < numTeachers - 1; i++)
	{
		teacherMap[i + 1] = js["teachers"][i];
	}
	for (int i = 0; i < numParents - 1; i++)
	{
		parentMap[i + 1] = js["parents"][i];
	}
	for (int i = 0; i < js["blockedTeacher"].size(); i++)
	{
		for (int j = 0; j < numTeachers; j++)
		{
			if (teacherMap[j + 1] == js["blockedTeacher"][i]["teacher"])
			{
				teachers[i] = j + 1;
				break;
			}
		}
		parents[i] = 0;
		singleTimes[i] = js["blockedTeacher"][i]["slot"];
	}
	for (int i = 0; i < js["blockedParent"].size(); i++)
	{
		for (int j = 0; j < numParents; j++)
		{
			if (parentMap[j + 1] == js["blockedParent"][i]["parent"])
			{
				parents[js["blockedTeacher"].size() + i] = j + 1;
				break;
			}
		}
		teachers[js["blockedTeacher"].size() + i] = 0;
		singleTimes[js["blockedTeacher"].size() + i] = js["blockedParent"][i]["slot"];
	}
	int *wantedTeachersTmp = (int *)malloc(wantedAppointments * sizeof(int) * 2);
	int *wantedParentsTmp = (int *)malloc(wantedAppointments * sizeof(int) * 2);
	for (int i = 0; i < wantedAppointments; i++)
	{
		for (int j = 0; j < numTeachers; j++)
		{
			if (teacherMap[j + 1] == js["wantedAppointments"][i]["teacher"])
			{
				wantedTeachersTmp[i] = j + 1;
				break;
			}
		}
		for (int j = 0; j < numParents; j++)
		{
			if (parentMap[j + 1] == js["wantedAppointments"][i]["parent"])
			{
				wantedParentsTmp[i] = j + 1;
				break;
			}
		}
		singleTimes[i + numBlocked] = 0;
	}
	// shuffle the wanted appointments
	// print the wanted appointments

	int *teachersTmp = (int *)malloc(js["appointments"].size() * sizeof(int) * 2);
	int *parentsTmp = (int *)malloc(js["appointments"].size() * sizeof(int) * 2);
	for (int i = 0; i < js["appointments"].size(); i++)
	{
		for (int j = 0; j < numTeachers - 1; j++)
		{
			if (teacherMap[j + 1] == js["appointments"][i]["teacher"])
			{
				teachersTmp[i] = j + 1;
				break;
			}
		}
		for (int j = 0; j < numParents - 1; j++)
		{
			if (parentMap[j + 1] == js["appointments"][i]["parent"])
			{
				parentsTmp[i] = j + 1;
				break;
			}
		}
		singleTimes[i + numBlocked + wantedAppointments] = 0;
	}
	for (int i = 0; i < numTeachers - 1; i++)
	{
		teacherMin[i + 1] = js["teacherMin"][i];
	}
	// teacher max
	for (int i = 0; i < numTeachers - 1; i++)
	{
		teacherMax[i + 1] = js["teacherMax"][i] - 1;
	}
	// parent min
	for (int i = 0; i < numParents - 1; i++)
	{
		parentMin[i + 1] = js["parentMin"][i];
	}
	// parent max
	for (int i = 0; i < numParents - 1; i++)
	{
		parentMax[i + 1] = js["parentMax"][i] - 1;
	}
	for (int Itr = 0; Itr < iterations; Itr++)
	{
		// set them all min and max to 0 and 16

		// fill teacher and parent arrays
		// loop through the blocked slots first
		shuffleSame(wantedTeachersTmp, wantedParentsTmp, wantedAppointments);

		// shuffle the appointments
		shuffleSame(teachersTmp, parentsTmp, numAppointments);
		// merge the wanted and normal appointments
		for (int i = 0; i < wantedAppointments; i++)
		{
			teachers[i + numBlocked] = wantedTeachersTmp[i];
			parents[i + numBlocked] = wantedParentsTmp[i];
		}
		for (int i = 0; i < numAppointments; i++)
		{
			teachers[i + numBlocked + wantedAppointments] = teachersTmp[i];
			parents[i + numBlocked + wantedAppointments] = parentsTmp[i];
		}
		// teacher min
		
		//store the parents and teachers arrays for the current iteration
		for (int i = 0; i < numAppointments + numBlocked + wantedAppointments; i++)
		{
			teachersStore[Itr * (numAppointments + numBlocked + wantedAppointments) + i] = teachers[i];
			parentsStore[Itr * (numAppointments + numBlocked + wantedAppointments) + i] = parents[i];
		}

		// allocate memory
		// duplicate singleTimes array

		int count = 0;
		for (int i = 0; i < numThreads; i++)
		{
			std::memcpy(&appTimes[i * (numAppointments + numBlocked + wantedAppointments)], singleTimes, (numAppointments + numBlocked + wantedAppointments) * sizeof(*singleTimes));
			count += (numAppointments + numBlocked + wantedAppointments);
		}
		// free memory

		hiprandState *d_States;
		hipMalloc(&d_States, numThreads * sizeof(hiprandState));

		int *d_teachers;
		hipMalloc(&d_teachers, (numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemcpy(d_teachers, teachers, (numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		int *d_teacherMin;
		hipMalloc(&d_teacherMin, numTeachers * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemcpy(d_teacherMin, teacherMin, numTeachers * sizeof(int) * 2, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		int *d_teacherMax;
		hipMalloc(&d_teacherMax, numTeachers * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemcpy(d_teacherMax, teacherMax, numTeachers * sizeof(int) * 2, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		int *d_appTimes;
		hipMalloc(&d_appTimes, numThreads * (numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemcpy(d_appTimes, appTimes, (numAppointments + numBlocked + wantedAppointments)* numThreads * sizeof(int) * 2, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		int *d_parents;
		hipMalloc(&d_parents, (numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemcpy(d_parents, parents, (numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		// print parents array

		// copy parent arrays to host
		hipMemcpy(parents, d_parents, (numAppointments + numBlocked + wantedAppointments) * sizeof(int) * 2, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		int *d_parentMin;
		hipMalloc(&d_parentMin, numParents * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemcpy(d_parentMin, parentMin, numParents * sizeof(int) * 2, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		int *d_parentMax;
		hipMalloc(&d_parentMax, numParents * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemcpy(d_parentMax, parentMax, numParents * sizeof(int) * 2, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		int *d_fitness;
		hipMalloc(&d_fitness, numThreads * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemset(d_fitness, 0, numThreads * sizeof(int) * 2);
		hipDeviceSynchronize();

		int *d_teacherTimes;
		hipMalloc(&d_teacherTimes, numThreads * numSlots * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemset(d_teacherTimes, 0, numThreads * numSlots * sizeof(int) * 2);
		hipDeviceSynchronize();

		int *d_parentTimes;
		hipMalloc(&d_parentTimes, numThreads * numSlots * sizeof(int) * 2);
		hipDeviceSynchronize();
		hipMemset(d_parentTimes, 0, numThreads * numSlots * sizeof(int) * 2);
		hipDeviceSynchronize();

		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			fprintf(stderr, "1 ERROR: %s\n", hipGetErrorString(error));
		}
		// generate random number
		srand(time(NULL) + Itr * 1000000);
		// rand int 1-1000000
		int seed = rand() % 1000000 + 1;
		initRand<<<blocksPerGrid, threadsPerBlock>>>(d_States, time(NULL) + Itr * 1000000 + seed);
		// print all the variables to check valid, numAppointments numThreads, legth of arrays etc
		// print legth of arrays

		// generate times

		bundledFunction<<<blocksPerGrid, threadsPerBlock, numThreads * sizeof(int)>>>(d_States, d_appTimes, d_teachers, d_teacherMin, d_teacherMax, numTeachers, d_parents, d_parentMin, d_parentMax, numParents, numSlots, numThreads, (numAppointments+ numBlocked + wantedAppointments), d_teacherTimes, d_parentTimes, numParents, numTeachers, d_fitness, numBlocked);
		hipDeviceSynchronize();
		// wait for the kernel to finish
		hipMemcpy(appTimes, d_appTimes, (numAppointments + numBlocked + wantedAppointments)* numThreads * sizeof(int) * 2, hipMemcpyDeviceToHost);
		// get errors
		error = hipGetLastError();
		if (error != hipSuccess)
		{
			fprintf(stderr, "3 ERROR: %s\n", hipGetErrorString(error));
		}

		// copy back fitnesses to host
		hipMemcpy(fitness, d_fitness, numThreads * sizeof(int) * 2, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		// print
		// find the best fitness
		int bestFitness = -2100000;
		int bestIndex = 0;
		for (int i = 0; i < numThreads; i++)
		{
			if (fitness[i] > bestFitness)
			{
				bestFitness = fitness[i];
				bestIndex = i;
			}
		}
		bestFitnesses[Itr] = bestFitness;
		// copy the best times to the best times array
		for (int i = 0; i < (numAppointments + numBlocked + wantedAppointments); i++)
		{
			bestTimes[Itr * (numAppointments + numBlocked + wantedAppointments) + i] = appTimes[bestIndex * (numAppointments + numBlocked + wantedAppointments) + i];
		}

		hipFree(d_States);
		hipFree(d_teachers);
		hipFree(d_teacherMin);
		hipFree(d_teacherMax);
		hipFree(d_appTimes);
		hipFree(d_parents);
		hipFree(d_parentMin);
		hipFree(d_parentMax);
		hipFree(d_fitness);
		hipFree(d_teacherTimes);
		hipFree(d_parentTimes);
	}
	// free memory
	free(teacherMin);
	free(teacherMax);
	free(parentMin);
	free(parentMax);
	free(fitness);
	free(appTimes);
	free(singleTimes);
	// print best time
	int bestFitness = -2100000;
	int bestIndex = 0;
	for (int i = 0; i < iterations; i++)
	{
		if (bestFitnesses[i] > bestFitness)
		{
			bestFitness = bestFitnesses[i];
			bestIndex = i;
		}
	}
	printf("Best Fitness: %d\n", bestFitness);
	for (int i = 0; i < (numAppointments + numBlocked + wantedAppointments); i++)
	{
		printf("%d ", bestTimes[bestIndex * (numAppointments + numBlocked + wantedAppointments) + i]);
	}

	std::string outputFileName = "output.json";
	std::ofstream outputFile(outputFileName);
	nlohmann::json outputJson;
	// to write this to a file the ids need to be converted back to the original ids, and the appointments can be found by the position in the parent and teacher arrays
	for (int i = 0; i < (numAppointments + numBlocked + wantedAppointments); i++)
	{
		int teacherId = teacherMap[teachersStore[bestIndex * (numAppointments + numBlocked + wantedAppointments) + i]];
		int parentId = parentMap[parentsStore[bestIndex * (numAppointments + numBlocked + wantedAppointments) + i]];
		outputJson["appointments"][i]["teacher"] = teacherId;
		outputJson["appointments"][i]["parent"] = parentId;
		outputJson["appointments"][i]["slot"] = bestTimes[bestIndex * (numAppointments + numBlocked + wantedAppointments) + i];
	}
	outputFile << outputJson.dump(4);
	outputFile.close();


	// free memory
	free(bestTimes);
	free(bestFitnesses);
	free(teachersStore);
	free(parentsStore);
	free(teachers);
	free(parents);
	free(wantedTeachersTmp);
	free(wantedParentsTmp);
	free(teachersTmp);
	free(parentsTmp);
	return 0;
}
